// nvcc vecadd.cu && ./a.out

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

// Compute vector sum C = A+B
// Each thread performs one pair-wise addition
__global__
void vecAddKernel(float* A, float* B, float* C, int n) {
    int i = (blockDim.x*blockIdx.x + threadIdx.x)*2;
    if(i<n) {
        C[i] = A[i] + B[i];
        int j = i + 1;
        if (j < n) {
            C[j] = A[j] + B[j];
        }
    }
}

void vecAdd(float* A, float* B, float* C, int n) {
    // kind of a stub function for launching a kernel
    hipError_t err;
    int size = n* sizeof(float);
    float *d_A, *d_B, *d_C;

    // Allocate device memory for A, B, and C
    err = hipMalloc((void **)&d_A, size);
    err = hipMalloc((void **)&d_B, size);
    err = hipMalloc((void **)&d_C, size);

    if (err !=hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
        return;
    }
    
    // copy A and B to device memory
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    // Kernel launch code – to have the device
    // to perform the actual vector addition
    vecAddKernel<<<ceil(n/64.0), 32>>>(d_A, d_B, d_C, n);
    // copy C from the device memory
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    // Free device vectors
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}


int main() {
    float A[100];
    float B[100];
    float C[100];

    for (int i = 0; i < 100; i ++) {
        A[i] = 1.0 * i;
        B[i] = 2.0 * i;
    }

    int n = sizeof(A)/sizeof(float);
    fprintf(stdout, "n: %d\n", n);

    vecAdd(A, B, C, n);

    fprintf(stdout, "all good!\n");
    for (int i = 0; i < 10; i ++) {
        fprintf(stdout, "C[%d]: %lf\n", i, C[i]);
    }
}
