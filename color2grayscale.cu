#include "hip/hip_runtime.h"
#include <stdio.h>
#include "lib/png.h"

#define RGB2Greyscale(r,g,b) (r*0.21f) + (g*0.72f) + (b*0.07f)
#define CHANNELS 4 // rgb + alpha

// we have 3 channels corresponding to RGB
// The input image is encoded as unsigned characters [0, 255]
__global__
void colorToGreyscaleConversion(
    unsigned char * Pout, unsigned char * Pin,
    int width, int height
) {
    int Col = threadIdx.x + blockIdx.x * blockDim.x;
    int Row = threadIdx.y + blockIdx.y * blockDim.y;
    if (Col < width && Row < height) {
        // get 1D coordinate for the grayscale image
        int greyOffset = Row*width + Col;
        // one can think of the RGB image having
        // CHANNEL times columns than the grayscale image
        int rgbOffset = greyOffset*CHANNELS;
        unsigned char r = Pin[rgbOffset ]; // red value for pixel
        unsigned char g = Pin[rgbOffset + 1]; // green value for pixel
        unsigned char b = Pin[rgbOffset + 2]; // blue value for pixel
        // perform the rescaling and store it
        // We multiply by floating point constants
        Pout[greyOffset] = RGB2Greyscale(r,g,b);
    }
}

int main(int argc, char *argv[]) {
    if (argc != 3) {
        fprintf(stderr, "Usage: %s <input.png> <grayscale_output.png>\n", argv[0]);
        return 1;
    }
    const char * filename = argv[1];
    const char * output_filename = argv[2];

    Image img = {0};

    if (!read_png_file(filename, &img)) {
        fprintf(stderr, "Failed to read PNG file.\n");
        return 1;
    }

    printf("Width: %u, Height: %u\n", img.width, img.height);

    // Example: Access first pixel (RGBA)
    printf("First pixel RGBA: %u %u %u %u\n",
           img.data[0], img.data[1], img.data[2], img.data[3]);

    int img_width = img.width;
    int img_height = img.height;
    int size = img_width*img_height;

    // check if all zero
    bool non_zero_found = false;
    for (int i = 0; !non_zero_found && i < size*CHANNELS; i ++) {
        if (img.data[i] > 0)
            non_zero_found = true;
    }
    if (non_zero_found) {
        fprintf(stderr, "[DEBUG] Non-zero pixels found in input image.\n");
    } else {
        fprintf(stderr, "[ERROR] No non-zero pixels found in input image\n");
    }

    unsigned char * out_png = (unsigned char *)malloc(size);

    unsigned char * d_Pin, * d_Pout;
    hipError_t err;

    // Allocate device memory for Pin and Pout
    err = hipMalloc((void **)&d_Pin, size*CHANNELS); // rgb
    err = hipMalloc((void **)&d_Pout, size);
    if (err !=hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
        return;
    }

     // copy Pin to device memory
    hipMemcpy(d_Pin, img.data, size*CHANNELS, hipMemcpyHostToDevice);

    // Kernel launch code – to have the device
    // to perform the actual vector addition
    dim3 dimGrid(ceil(img_width/16.0), ceil(img_height/16.0), 1);
    dim3 dimBlock(16, 16, 1);
    colorToGreyscaleConversion<<<dimGrid,dimBlock>>>(d_Pout, d_Pin, img_width, img_height);

    // copy Pout from the device memory
    hipMemcpy(out_png, d_Pout, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; i ++) {
        fprintf(stdout, "%d\t", out_png[i]);
    } printf("\n");

    // check if all zero
    non_zero_found = false;
    for (int i = 0; !non_zero_found && i < size; i ++) {
        if (out_png[i] > 0)
            non_zero_found = true;
    }
    if (non_zero_found) {
        fprintf(stderr, "[DEBUG] Non-zero pixels found in output image.\n");
    } else {
        fprintf(stderr, "[ERROR] No non-zero pixels found in output image\n");
    }

    // Free device vectors
    hipFree(d_Pin);
    hipFree(d_Pout);

    write_png_grayscale(output_filename, out_png, img_width, img_height);
    
    // Free host memory
    free(img.data);
    free(out_png);
}
